﻿#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cmath>
#include <fstream>
#include <iomanip>
#include <sstream>
#include <chrono>


__device__ void Q2F(double* Q, double* F, int idx, double gamma) {
    double rho = Q[idx * 4];
    double u = Q[idx * 4 + 1] / rho;
    double v = Q[idx * 4 + 2] / rho;
    double E = Q[idx * 4 + 3];
    double p = (gamma - 1.0) * (E - 0.5 * rho * (u * u + v * v));

    F[0] = rho * u;
    F[1] = rho * u * u + p;
    F[2] = rho * u * v;
    F[3] = u * (E + p);
}

__device__ void Q2G(double* Q, double* G, int idx, double gamma) {
    double rho = Q[idx * 4];
    double u = Q[idx * 4 + 1] / rho;
    double v = Q[idx * 4 + 2] / rho;
    double E = Q[idx * 4 + 3];
    double p = (gamma - 1.0) * (E - 0.5 * rho * (u * u + v * v));

    G[0] = rho * v;
    G[1] = rho * u * v;
    G[2] = rho * v * v + p;
    G[3] = v * (E + p);
}

// Kernel to update the solution
__global__ void update_solution(double* Q, double* Q_new, int nx, int ny, double dx, double dy, double dt, double gamma) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i > 0 && i < nx - 1 && j > 0 && j < ny - 1) {
        int idx = i + j * nx;
        int idx_ip1 = (i + 1) + j * nx;
        int idx_im1 = (i - 1) + j * nx;
        int idx_jp1 = i + (j + 1) * nx;
        int idx_jm1 = i + (j - 1) * nx;

        double F_ip1[4], F_im1[4], G[4], G_jp1[4], G_jm1[4];
        Q2F(Q, F_ip1, idx_ip1, gamma);
        Q2F(Q, F_im1, idx_im1, gamma);
        Q2G(Q, G_jp1, idx_jp1, gamma);
        Q2G(Q, G_jm1, idx_jm1, gamma);

        // Lax-Friedrichs update
        for (int k = 0; k < 4; k++) {
            Q_new[idx * 4 + k] = 0.25 * (Q[idx_ip1 * 4 + k] + Q[idx_im1 * 4 + k] + Q[idx_jp1 * 4 + k] + Q[idx_jm1 * 4 + k])
                - dt / (2 * dx) * (F_ip1[k] - F_im1[k])
                - dt / (2 * dy) * (G_jp1[k] - G_jm1[k]);
        }
    }
}

// Kernel to apply boundary conditions
__global__ void apply_boundary_conditions(double* Q, int nx, int ny) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Apply boundary conditions at the left and right boundaries
    if (i == 0) {
        for (int k = 0; k < ny; k++) {
            int idx = i + k * nx;
            Q[idx * 4 + 1] = 0.0; // Set horizontal velocity (rho*u) to 0
            // Other are equal
            Q[idx * 4 + 0] = Q[(i + 1 + k * nx) * 4 + 0];
            Q[idx * 4 + 2] = Q[(i + 1 + k * nx) * 4 + 2];
            Q[idx * 4 + 3] = Q[(i + 1 + k * nx) * 4 + 3];
        }
    }

    // Apply boundary conditions at the top and bottom boundaries
    if (j == 0) {
        for (int k = 0; k < nx; k++) {
            int idx = k + j * nx;
            Q[idx * 4 + 2] = 0.0; // Set vertical velocity (rho*v) to 0
            // Other are equal
            Q[idx * 4 + 0] = Q[(k + (j + 1) * nx) * 4 + 0];
            Q[idx * 4 + 1] = Q[(k + (j + 1) * nx) * 4 + 1];
            Q[idx * 4 + 3] = Q[(k + (j + 1) * nx) * 4 + 3];
        }
    }

    // Apply boundary conditions at the left and right boundaries
    if (i == nx - 1) {
        for (int k = 0; k < ny; k++) {
            int idx = i + k * nx;
            Q[idx * 4 + 1] = 0.0; // Set horizontal velocity (rho*u) to 0
            // Other are equal
            Q[idx * 4 + 0] = Q[(i - 1 + k * nx) * 4 + 0];
            Q[idx * 4 + 2] = Q[(i - 1 + k * nx) * 4 + 2];
            Q[idx * 4 + 3] = Q[(i - 1 + k * nx) * 4 + 3];
        }
    }

    // Apply boundary conditions at the top and bottom boundaries
    if (j == ny - 1) {
        for (int k = 0; k < nx; k++) {
            int idx = k + j * nx;
            Q[idx * 4 + 2] = 0.0; // Set vertical velocity (rho*v) to 0
            // Other are equal
            Q[idx * 4 + 0] = Q[(k + (j - 1) * nx) * 4 + 0];
            Q[idx * 4 + 1] = Q[(k + (j - 1) * nx) * 4 + 1];
            Q[idx * 4 + 3] = Q[(k + (j - 1) * nx) * 4 + 3];
        }
    }
}

// Kernel to compute the maximum wave speed
__global__ void compute_max_wave_speed(double* Q, double* max_wave_speed, int nx, int ny, double gamma) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < nx && j < ny) {
        int idx = i + j * nx;

        double rho = Q[idx * 4 + 0];
        double u = Q[idx * 4 + 1] / rho;
        double v = Q[idx * 4 + 2] / rho;
        double E = Q[idx * 4 + 3];
        double p = (gamma - 1.0) * (E - 0.5 * rho * (u * u + v * v));
        double c = sqrt(gamma * p / rho);

        max_wave_speed[idx] = fabs(u) + c + fabs(v) + c;
    }
}


// Function to write data to a VTK file
void write_vtk(const std::vector<double>& Q, int nx, int ny, int timestep, double dx, double dy, double gamma) {
    std::ostringstream filename;
    filename << "output_" << std::setw(4) << std::setfill('0') << timestep << ".vtk";
    std::ofstream file(filename.str());
    std::vector<double> density(Q.size()/4);
    std::vector<double> velocity_x(Q.size() / 4);
    std::vector<double> velocity_y(Q.size() / 4);
    std::vector<double> pressure(Q.size() / 4);

    for (int idx = 0; idx < Q.size() / 4; idx++) {
        density[idx] = Q[idx * 4];
        velocity_x[idx] = Q[idx * 4 + 1]/density[idx];
        velocity_y[idx] = Q[idx * 4 + 2]/density[idx];
        pressure[idx] = (Q[idx * 4 + 3] - 0.5 * density[idx] * (velocity_x[idx] * velocity_x[idx] + velocity_y[idx] * velocity_y[idx])) * (gamma - 1.);
    }


    file << "# vtk DataFile Version 2.0\n";
    file << "Euler Equations\n";
    file << "ASCII\n";
    file << "DATASET STRUCTURED_GRID\n";
    file << "DIMENSIONS " << nx << " " << ny << " 1\n";
    file << "POINTS " << nx * ny << " double\n";

    for (int j = 0; j < ny; j++) {
        for (int i = 0; i < nx; i++) {
            file << i * dx << " " << j * dy << " 0\n";
        }
    }

    file << "POINT_DATA " << nx * ny << "\n";

    file << "SCALARS density double 1\n";
    file << "LOOKUP_TABLE default\n";
    for (int j = 0; j < ny; j++) {
        for (int i = 0; i < nx; i++) {
            int idx = i + j * nx;
            file << std::fixed << density[idx + 0] << "\n";
        }
    }

    file << "SCALARS velocity_x double 1\n";
    file << "LOOKUP_TABLE default\n";
    for (int j = 0; j < ny; j++) {
        for (int i = 0; i < nx; i++) {
            int idx = i + j * nx;
            file << std::fixed << velocity_x[idx] << "\n";
        }
    }

    file << "SCALARS velocity_y double 1\n";
    file << "LOOKUP_TABLE default\n";
    for (int j = 0; j < ny; j++) {
        for (int i = 0; i < nx; i++) {
            int idx = i + j * nx;
            file << std::fixed << velocity_y[idx] << "\n";
        }
    }

    file << "SCALARS pressure double 1\n";
    file << "LOOKUP_TABLE default\n";
    for (int j = 0; j < ny; j++) {
        for (int i = 0; i < nx; i++) {
            int idx = i + j * nx;
            file << std::fixed << pressure[idx] << "\n";
        }
    }

    file.close();
}

void cyl_Riemann(std::vector<double>& initial, int nx, int ny, double dx, double dy, double gamma) {
    const std::pair<double, double> center = { 1., 1. };
    const double radius = 0.3;
    for (int i = 0; i < nx; i++) {
        for (int j = 0; j < ny; j++) {
            int idx = i + j * nx;
            if ((i * dx - center.first) * (i * dx - center.first) + (j * dy - center.second) * (j * dy - center.second) < radius) {
                initial[idx * 4 + 0] = 1.0;  // rho
                initial[idx * 4 + 1] = 0.0;  // rho*u
                initial[idx * 4 + 2] = 0.0;  // rho*v
                initial[idx * 4 + 3] = 1.0 / (gamma - 1.);  // E
            }
            else {
                initial[idx * 4 + 0] = 0.125;  // rho
                initial[idx * 4 + 1] = 0.0;  // rho*u
                initial[idx * 4 + 2] = 0.0;  // rho*v
                initial[idx * 4 + 3] = 0.1 / (gamma - 1.);  // E
            }
        }
    }
}

int main() {
    // Constants
    const int NX = 2001;  // Number of grid points in x direction
    const int NY = 2001;  // Number of grid points in y direction
    const double gamma = 1.4;  // Ratio of specific heats
    const double dx = 2.0 / (NX - 1);  // Grid spacing in x direction
    const double dy = 2.0 / (NY - 1);  // Grid spacing in y direction
    const double cfl_number = 0.75;  // CFL number

    // Allocate host memory
    std::vector<double> Q(NX * NY * 4);
    std::vector<double> Q_new(NX * NY * 4);
    std::vector<double> max_wave_speed(NX * NY);

    // Initialize Q (this would typically involve setting initial conditions)
    //for (int j = 0; j < NY; j++) {
    //    for (int i = 0; i < NX; i++) {
    //        int idx = i + j * NX;
    //        if (i * dx < 1.) {
    //            Q[idx * 4 + 0] = 1.0;  // rho
    //            Q[idx * 4 + 1] = 0.0;  // rho*u
    //            Q[idx * 4 + 2] = 0.0;  // rho*v
    //            Q[idx * 4 + 3] = 1.0 / (gamma - 1.);  // E
    //        }
    //        else {
    //            Q[idx * 4 + 0] = 0.125;  // rho
    //            Q[idx * 4 + 1] = 0.0;  // rho*u
    //            Q[idx * 4 + 2] = 0.0;  // rho*v
    //            Q[idx * 4 + 3] = 0.1 / (gamma - 1.);  // E
    //        }        
    //    }
    //}

    cyl_Riemann(Q, NX, NY, dx, dy, gamma);



    // Allocate device memory
    double* d_Q, * d_Q_new, * d_max_wave_speed;
    hipMalloc(&d_Q, NX * NY * 4 * sizeof(double));
    hipMalloc(&d_Q_new, NX * NY * 4 * sizeof(double));
    hipMalloc(&d_max_wave_speed, NX * NY * sizeof(double));

    // Copy data from host to device
    hipMemcpy(d_Q, Q.data(), NX * NY * 4 * sizeof(double), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((NX + blockDim.x - 1) / blockDim.x, (NY + blockDim.y - 1) / blockDim.y);

    double Time = 0.0;       // Time
    double Time_end = 0.75;  // End
    int n = 0;               // Counter

    double another_time = 0.0;

    auto begin = std::chrono::steady_clock::now();

    while (Time < Time_end) {
        // Increase counter
        //if (Time >= another_time) {
        //    write_vtk(Q, NX, NY, n++, dx, dy, gamma);
        //    another_time += 0.01;
        //}

        // CFL Control
        compute_max_wave_speed <<<gridDim, blockDim >>> (d_Q, d_max_wave_speed, NX, NY, gamma);
        hipMemcpy(max_wave_speed.data(), d_max_wave_speed, NX * NY * sizeof(double), hipMemcpyDeviceToHost);
        double max_wave_speed_host = *std::max_element(max_wave_speed.begin(), max_wave_speed.end());
        double dt = cfl_number * std::min(dx, dy) / max_wave_speed_host;

        // Launch kernel to update solution
        update_solution <<<gridDim, blockDim >>> (d_Q, d_Q_new, NX, NY, dx, dy, dt, gamma);

        // Swap pointers
        std::swap(d_Q, d_Q_new);

        // Apply boundary conditions
        apply_boundary_conditions <<<gridDim, blockDim >>> (d_Q, NX, NY);

        // Copy data from device to host for output
        hipMemcpy(Q.data(), d_Q, NX * NY * 4 * sizeof(double), hipMemcpyDeviceToHost);

        // Write data to VTK file
        //write_vtk(Q, NX, NY, n, dx, dy, gamma);

        // Increase time
        Time += dt;

        std::cout << "Time = " << Time << std::endl;
    }

    auto end = std::chrono::steady_clock::now();
    std::cout << "Time difference = " << std::chrono::duration_cast<std::chrono::microseconds>(end - begin).count() << "[µs]" << std::endl;

    std::cout << "All steps = " << n << std::endl;


    //write_vtk(Q, NX, NY, 0, dx, dy, gamma);


    // Copy data from device to host
    hipMemcpy(Q.data(), d_Q, NX * NY * 4 * sizeof(double), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_Q);
    hipFree(d_Q_new);
    hipFree(d_max_wave_speed);

    return 0;
}